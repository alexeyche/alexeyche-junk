#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>


__global__ void run(unsigned char *source, int width, int height, unsigned char *destination) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    float sum = 0.0;

    // int   pWidth = kWidth/2;
    // int   pHeight = kHeight/2;

    // // Only execute for valid pixels
    // if(x >= pWidth+paddingX &&
    //    y >= pHeight+paddingY &&
    //    x < (blockDim.x * gridDim.x)-pWidth-paddingX &&
    //    y < (blockDim.y * gridDim.y)-pHeight-paddingY)
    // {
    //     for(int j = -pHeight; j <= pHeight; j++)
    //     {
    //         for(int i = -pWidth; i <= pWidth; i++)
    //         {
    //             // Sample the weight for this location
    //             int ki = (i+pWidth);
    //             int kj = (j+pHeight);
    //             float w  = convolutionKernelStore[(kj * kWidth) + ki + kOffset];

        
    //             sum += w * float(source[((y+j) * width) + (x+i)]);
    //         }
    //     }
    // }
    
    destination[(y * width) + x] = (unsigned char) sum;
}

// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
void* createImageBuffer(unsigned int bytes, void **devicePtr)
{
    void *ptr = NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
    hipHostGetDevicePointer(devicePtr, ptr, 0);
    return ptr;
}





int main (int argc, char** argv)
{
    // Open a webcamera
    cv::VideoCapture camera(0);
    cv::Mat          frame;
    if(!camera.isOpened()) 
        return -1;
    
    // Create the capture windows
    cv::namedWindow("Source");
    cv::namedWindow("Greyscale");
    cv::namedWindow("Dest");
    
    // Create the cuda event timers 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    camera >> frame;
    
    unsigned char *sourceDevice;
    cv::Mat source  (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, (void**)(&sourceDevice)));
    
    size_t ncHeight = frame.size().height/10;
    size_t ncWidth = frame.size().width/10;
    size_t ncNum = ncWidth * ncHeight;
    cv::Size ncSize(ncWidth, ncHeight);

    float *weightsDevice;
    cv::Mat weights(ncSize, CV_32F, createImageBuffer(sizeof(float) * ncNum, (void**)&weightsDevice));
    
    
    unsigned char *destDevice;
    cv::Mat dest(frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, (void**)(&destDevice)));
    
    while(1) {
        
        camera >> frame;
        cv::cvtColor(frame, source, CV_BGR2GRAY);
        
        hipEventRecord(start);
        {
            dim3 cblocks (ncHeight / 4, ncWidth / 4);
            dim3 cthreads(4, 4);

            
            run<<<cblocks,cthreads>>>(sourceDevice, frame.size().width, frame.size().height, destDevice);
            
            hipDeviceSynchronize();
        }
        hipEventRecord(stop);
        
        float ms = 0.0f;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        std::cout << "Elapsed GPU time: " << ms << " milliseconds" << std::endl;

        // Show the results
        cv::imshow("Source", frame);
        cv::imshow("Greyscale", source);
        cv::imshow("Dest", dest);
        
        // Spin
        if(cv::waitKey(1) == 27) break;
    }
    
    // Exit
    hipHostFree(source.data);
    hipHostFree(dest.data);
    hipFree(weights.data);    
    return 0;
}
